
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__global__ 
void vecAddKernel(float* A, float* B, float* C, int n) 
{

    // Calculate global thread index based on the block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use global index to determine which elements to read, add, and write
    if(i < n)
    {
        C[i] = A[i] + B[i];
    }

}

